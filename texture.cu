
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <iostream>

//typedef uint8_t float;  // use an integer type

const int num_rows = 4;
const int num_cols = 4;

__global__ void kernel(hipTextureObject_t tex, float xMax, float yMax)
{
  float offset = 0.5;
  float xScale = 1. / xMax;
  float yScale = 1. / xMax;

  printf("No offset \n");
  for (int ii = 0; ii < num_rows; ++ii) {
    for (int jj = 0; jj < num_cols; ++jj) {
      float x = (float)jj;
      float y = (float)ii;
      float val = tex2D<float>(tex, (x+offset)*xScale, (y+offset)*yScale);
      printf("%.2f, ", val);
    }
    printf("\n");
  }
  printf("\n");
  
  printf("X offset (half bin)\n");
  for (int ii = 0; ii < num_rows; ++ii) {
    for (int jj = 0; jj < num_cols; ++jj) {
      float x = (float)jj+0.5;
      float y = (float)ii;
      float val = tex2D<float>(tex, (x+offset)*xScale, (y+offset)*yScale);
      printf("%.2f, ", val);
    }
    printf("\n");
  }
  printf("\n");

  printf("Y offset (half bin)\n");
  for (int ii = 0; ii < num_rows; ++ii) {
    for (int jj = 0; jj < num_cols; ++jj) {
      float x = (float)jj;
      float y = (float)ii+0.5;
      float val = tex2D<float>(tex, (x+offset)*xScale, (y+offset)*yScale);
      printf("%.2f, ", val);
    }
    printf("\n");
  }
  printf("\n");

  printf("X and Y offset (half bin)\n");
  for (int ii = 0; ii < num_rows; ++ii) {
    for (int jj = 0; jj < num_cols; ++jj) {
      float x = (float)jj+0.5;
      float y = (float)ii+0.5;
      float val = tex2D<float>(tex, (x+offset)*xScale, (y+offset)*yScale);
      printf("%.2f, ", val);
    }
    printf("\n");
  }
  printf("\n");
}

//const int num_cols = prop.texturePitchAlignment*1; // should be able to use a different multiplier here

int main(int argc, char **argv)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("texturePitchAlignment: %lu\n", prop.texturePitchAlignment);

    float xMax = num_cols;
    float yMax = num_rows;
    
    hipTextureObject_t tex;
    float dataIn[num_cols*num_rows*sizeof(float)];
    std::cout << "Input" << std::endl;
    for (int ii = 0; ii < num_rows; ii++) {
      for (int jj = 0; jj < num_cols; jj++) {
        int index = (ii*num_cols)+jj;
        dataIn[index] = (float)index*1.5;
        std::cout << dataIn[index] << " ";
      }
      std::cout << std::endl;
    }
    std::cout << std::endl;
    
    float* dataDev = 0;
    size_t pitch;
    
    hipMallocPitch((void**)&dataDev, &pitch,  num_cols*sizeof(float), num_rows);
    hipMemcpy2D(dataDev, pitch, dataIn, num_cols*sizeof(float), num_cols*sizeof(float), num_rows, hipMemcpyHostToDevice);
    
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dataDev;
    resDesc.res.pitch2D.width = num_cols;
    resDesc.res.pitch2D.height = num_rows;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
    resDesc.res.pitch2D.pitchInBytes = pitch;
    
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = true;

    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    dim3 threads(1, 1);
    kernel<<<1, threads>>>(tex, xMax, yMax);
    hipDeviceSynchronize();
    printf("\n");
    return 0;
}
