#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

typedef float4 mt;  // use an integer type

__global__ void kernel(hipTextureObject_t tex)
{
    float x = 0.5;
    float y = 0.5;
    mt val = tex2D<mt>(tex, x, y);
    printf("%f, ", val.x);
}

int main(int argc, char **argv)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("texturePitchAlignment: %lu\n", prop.texturePitchAlignment);
    hipTextureObject_t tex;
    const int num_rows = 4;
    const int num_cols = prop.texturePitchAlignment*1; // should be able to use a different multiplier here
    const int ts = num_cols*num_rows;
    const int ds = ts*sizeof(mt);
    mt dataIn[ds];
    for (int i = 0; i < ts; i++) dataIn[i] = float4(0,1,2,3);
    mt* dataDev = 0;
    hipMalloc((void**)&dataDev, ds);
    hipMemcpy(dataDev, dataIn, ds, hipMemcpyHostToDevice);
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dataDev;
    resDesc.res.pitch2D.width = num_cols;
    resDesc.res.pitch2D.height = num_rows;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<mt>();
    resDesc.res.pitch2D.pitchInBytes = num_cols*sizeof(mt);
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    dim3 threads(4, 4);
    kernel<<<1, 1>>>(tex);
    hipDeviceSynchronize();
    printf("\n");
    return 0;
}
